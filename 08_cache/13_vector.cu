
#include <hip/hip_runtime.h>
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <hipblas.h>
#include <chrono>
using namespace std;

__global__ void kernel(int dim_m, int dim_n, int dim_k,
		       float *d_a, float *d_b, float *d_c) {
  int offset_a_m = 64 * blockIdx.x;
  int offset_b_n = 64 * blockIdx.y;
  int a_m = threadIdx.x % 8 * 8;
  int a_k = threadIdx.x / 8;
  int b_k = 0;
  int b_n = threadIdx.x;

  struct __align__(16) vec_t { float d[8]; };
  __shared__ float __align__(16) block_a[8][64];
  __shared__ float __align__(16) block_b[8][64];
  float __align__(16) block_c[8][8];

  vec_t *tile_a = reinterpret_cast<vec_t*>(&d_a[a_k * dim_m + (offset_a_m + a_m)]);
  vec_t *tile_b = reinterpret_cast<vec_t*>(&d_b[(offset_b_n + b_n) * dim_k + b_k]);
  for (int m = 0; m < 8; ++m)
    for (int n = 0; n < 8; ++n)
      block_c[m][n] = 0;

  int offset_n = threadIdx.x / 8 * 8;
  int offset_m = threadIdx.x % 8 * 8;
  int offset_a_k = 0;
  int offset_b_k = 0;
  for (int k = 0; k < dim_k; k += 8) {
    vec_t __align__(16) thread_a = tile_a[offset_a_k];
    vec_t __align__(16) thread_b = tile_b[offset_b_k];
    __syncthreads();
    for (int j = 0; j < 8; ++j) {
      block_a[a_k][a_m + j] = thread_a.d[j];
      block_b[j][b_n] = thread_b.d[j];
    }
    __syncthreads();
    offset_a_k += dim_m;
    offset_b_k++;
#pragma unroll
    for (int k = 0; k < 8; ++k) {
      for (int m = 0; m < 8; ++m) {
	for (int n = 0; n < 8; ++n) {
	  block_c[m][n] += block_a[k][offset_m + m] * block_b[k][offset_n + n];
	}
      }
    }
  }
  for (int n = 0; n < 8; ++n) {
    for (int m = 0; m < 8; ++m) {
      int c_n = offset_b_n + offset_n + n;
      int c_m = offset_a_m + offset_m + m;
      if (c_n < dim_n && c_m < dim_m) {
	d_c[c_n * dim_m + c_m] = block_c[m][n];
      }
    }
  }
}

int main(int argc, const char **argv) {
  int m = 10240;
  int k = 4096;
  int n = 8192;
  float alpha = 1.0;
  float beta = 0.0;
  int Nt = 10;
  float *A, *B, *C, *C2;
  hipMallocManaged(&A, m * k * sizeof(float));
  hipMallocManaged(&B, k * n * sizeof(float));
  hipMallocManaged(&C, m * n * sizeof(float));
  hipMallocManaged(&C2, m * n * sizeof(float));
  for (int i=0; i<m; i++)
    for (int j=0; j<k; j++)
      A[k*i+j] = drand48();
  for (int i=0; i<k; i++)
    for (int j=0; j<n; j++)
      B[n*i+j] = drand48();
  for (int i=0; i<n; i++)
    for (int j=0; j<m; j++)
      C[m*i+j] = C2[m*i+j] = 0;
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  auto tic = chrono::steady_clock::now();
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    hipblasSgemm(cublas_handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		m,
		n,
		k,
		&alpha,
		A,
		m,
		B,
		k,
		&beta,
		C,
		m);
    hipDeviceSynchronize();
  }
  auto toc = chrono::steady_clock::now();
  int64_t num_flops = (2 * int64_t(m) * int64_t(n) * int64_t(k)) + (2 * int64_t(m) * int64_t(n));
  double tcublas = chrono::duration<double>(toc - tic).count() / Nt;
  double cublas_flops = double(num_flops) / tcublas / 1.0e9;
  int tile = 64;
  dim3 block = dim3(tile);
  dim3 grid = dim3((m+tile-1)/tile, (n+tile-1)/tile);
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    kernel<<< grid, block >>>(m,
			      n,
			      k,
			      A,
			      B,
			      C2);
    hipDeviceSynchronize();
  }
  toc = chrono::steady_clock::now();
  double tcutlass = chrono::duration<double>(toc - tic).count() / Nt;
  double cutlass_flops = double(num_flops) / tcutlass / 1.0e9;
  printf("CUBLAS: %.2f Gflops, CUTLASS: %.2f Gflops\n", cublas_flops, cutlass_flops);
  double err = 0;
  for (int i=0; i<n; i++) {
    for (int j=0; j<m; j++) {
      err += fabs(C[m*i+j] - C2[m*i+j]);
    }
  }
  printf("error: %lf\n", err/n/m);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(C2);
  hipblasDestroy(cublas_handle);
}
